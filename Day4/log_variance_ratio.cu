#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

// Matrix dimensions
const int nb_genes = 10000; // number of gens (columns)
const int nb_samples = 80; // number of samples (rows)


int threads_per_block = 128;
int num_pairs = (nb_genes * (nb_genes - 1)) / 2;
int num_blocks = num_pairs;

#define BLOCK_SIZE 128

__global__
void computeLogRatioVariance(float *d_Y, float *d_variances, int nb_samples, int nb_genes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // gene i
    int j = blockIdx.y * blockDim.y + threadIdx.y; // gene j

    if (i < nb_genes && j < i) {
        float sum = 0.0f;
        float sumsq = 0.0f;
        int k = 0;

        // Process 4 samples at a time using vector loads
        for (; k <= nb_samples - 4; k += 4) {
            // Load 4 elements for gene i and j using float4
            float4 y_i = *reinterpret_cast<float4*>(&d_Y[k + i * nb_samples]);
            float4 y_j = *reinterpret_cast<float4*>(&d_Y[k + j * nb_samples]);
            // Directly access the components (x, y, z, w)
            float ratio[4] = {y_i.x / y_j.x,y_i.y / y_j.y,y_i.z / y_j.z, y_i.w / y_j.w};
            for (int m = 0; m < 4; ++m) {
                float log_val = logf(ratio[m]);
                sum += log_val;
                sumsq += log_val * log_val;
            }
        }

        // Process remaining samples (0-3)
        for (; k < nb_samples; ++k) {
            float yi = d_Y[k + i * nb_samples];
            float yj = d_Y[k + j * nb_samples];
            float ratio = yi / yj;
            float log_val = logf(ratio);
            sum += log_val;
            sumsq += log_val * log_val;
        }

        // Compute variance
        float mean = sum / nb_samples;
        float variance = (sumsq - nb_samples * mean * mean) / (nb_samples - 1);
        int pair_index = (i * (i - 1)) / 2 + j;
        d_variances[pair_index] = variance;
    }
}


// CPU implementation for log variance ratio benchmark
float* compute_log_variance_ratio_cpu(const float* Y, int nb_samples, int nb_genes) {
    // Output array to store variances for each pair
    int num_pairs = (nb_genes * (nb_genes - 1)) / 2;
    float* variances = new float[num_pairs];
    int counter = 0;

    // For each pair of genes
    for(int i = 1; i < nb_genes; i++) {
        for(int j = 0; j < i; j++) {
            float mean = 0.0f;
            float variance = 0.0f;
            
            // First pass: compute mean of log ratios
            for(int k = 0; k < nb_samples; k++) {
                float ratio = Y[k + i * nb_samples] / Y[k + j * nb_samples];
                mean += log(ratio);
            }
            mean /= nb_samples;
            
            // Second pass: compute variance
            for(int k = 0; k < nb_samples; k++) {
                float ratio = Y[k + i * nb_samples] / Y[k + j * nb_samples];
                float diff = log(ratio) - mean;
                variance += diff * diff;
            }
            
            // Divide by (N-1) for sample variance
            variances[counter] = variance / (nb_samples - 1);
            counter++;
        }
    }

    return variances;
}

struct PerformanceMetrics {
    float kernel_time;      // milliseconds
    float memory_time;      // milliseconds
    float total_time;       // milliseconds
    float gflops;          // Floating point operations per second
    float bandwidth;       // GB/s
};

void initializeMatrice(float* Y, int nb_samples, int nb_genes) {
    for(int i = 0; i < nb_samples * nb_genes; i++) Y[i] = rand() / (float)RAND_MAX;
}

bool verifyResults(float* variances_gpu, float* variances_cpu, int num_pairs) {
    const float epsilon = 1e-2;
    for(int i = 0; i < num_pairs; i++) {
        if(abs(variances_gpu[i] - variances_cpu[i]) > epsilon) {
            printf("Verification failed at index %d: GPU=%f, CPU=%f\n", i, variances_gpu[i], variances_cpu[i]);
            return false;
        }
    }
    return true;
}


PerformanceMetrics benchmarkLogVarianceRatio() {
    PerformanceMetrics metrics;
    
    // Allocate host memory
    float *h_Y = (float*)malloc(nb_samples * nb_genes * sizeof(float));
    float *h_variances_cpu = (float*)malloc(num_pairs * sizeof(float));
    float *h_variances_gpu = (float*)malloc(num_pairs * sizeof(float));
    
    // Initialize matrices
    initializeMatrice(h_Y, nb_samples, nb_genes);
    
    // Allocate device memory
    float *d_Y, *d_variances_gpu;
    hipMalloc(&d_Y, nb_samples * nb_genes * sizeof(float));
    hipMalloc(&d_variances_gpu, num_pairs * sizeof(float));
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Memory transfer timing
    hipEventRecord(start);
    hipMemcpy(d_Y, h_Y, nb_samples * nb_genes * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&metrics.memory_time, start, stop);
    
    hipEventRecord(start);
    dim3 blockDim(8, 8);
    dim3 gridDim((nb_genes + blockDim.x - 1) / blockDim.x, (nb_genes + blockDim.y - 1) / blockDim.y);
    computeLogRatioVariance<<<gridDim, blockDim>>>(d_Y, d_variances_gpu, nb_samples, nb_genes);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&metrics.kernel_time, start, stop);
    
    // Copy result back
    hipMemcpy(h_variances_gpu, d_variances_gpu, num_pairs * sizeof(float), hipMemcpyDeviceToHost);
    
    // Calculate performance metrics
    metrics.total_time = metrics.kernel_time + metrics.memory_time;
    float operations = 2.0f * nb_samples * nb_genes * nb_genes;  // multiply-add per element
    metrics.gflops = (operations / 1e9) / (metrics.kernel_time / 1000.0f);
    metrics.bandwidth = (3.0f * nb_samples * nb_genes * sizeof(float)) / (metrics.total_time * 1e6);  // GB/s
    
    // Verify results
    h_variances_cpu = compute_log_variance_ratio_cpu(h_Y, nb_samples, nb_genes);
    bool correct = verifyResults(h_variances_gpu, h_variances_cpu, num_pairs);
    
    // Print performance metrics
    printf("\nPerformance Metrics:\n");
    printf("Matrix Size: %dx%d\n", nb_samples, nb_genes);
    printf("  +-- Kernel Time:     %.2f ms\n", metrics.kernel_time);
    printf("  +-- Memory Time:     %.2f ms\n", metrics.memory_time);
    printf("Total Time: %.3f ms\n", metrics.total_time);
    printf("Performance: %.2f GFLOPs\n", metrics.gflops);
    printf("Memory Bandwidth: %.2f GB/s\n", metrics.bandwidth);
    printf("Results: %s\n", correct ? "PASSED" : "FAILED");
    
    // Cleanup
    free(h_Y); free(h_variances_cpu); free(h_variances_gpu);
    hipFree(d_Y);
    hipFree(d_variances_gpu);
    
    return metrics;
}

int main() {
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max threads in X-dimension: %d\n", prop.maxThreadsDim[0]);
    
    // Run benchmark
    PerformanceMetrics metrics = benchmarkLogVarianceRatio();

    // print the metrics to console 
    printf("\n=== Log Variance Ratio Benchmark Report ===\n");
    printf("============================================\n");
    printf("Performance Summary:\n");
    printf("--------------------------------------------\n");
    printf("Total Execution Time: %.2f ms\n", metrics.total_time);
    printf("  +-- Kernel Time:     %.2f ms\n", metrics.kernel_time); 
    printf("  +-- Memory Time:     %.2f ms\n", metrics.memory_time);
    printf("\nCompute Performance:\n");
    printf("--------------------------------------------\n");
    printf("GFLOP/s:             %.2f\n", metrics.gflops);
    printf("Memory Bandwidth:     %.2f GB/s\n", metrics.bandwidth);
    printf("============================================\n");

    // Run CPU benchmark for comparison
    float cpu_time;
    {
        float *h_Y = (float*)malloc(nb_samples * nb_genes * sizeof(float));
        float *h_variances_cpu = (float*)malloc(num_pairs * sizeof(float));
        
        initializeMatrice(h_Y, nb_samples, nb_genes);
        
        auto start_time = clock();
        compute_log_variance_ratio_cpu(h_Y, nb_samples, nb_genes);
        auto end_time = clock();
        
        cpu_time = (float)(end_time - start_time) / CLOCKS_PER_SEC * 1000.0f; // Convert to ms
        
        free(h_Y);
        free(h_variances_cpu);
    }
    
    // Calculate CPU metrics
    float cpu_gflops = (2.0f * nb_samples * nb_genes * nb_genes) / (cpu_time * 1e6);
    
    printf("\n=== CPU vs GPU Comparison ===\n");
    printf("--------------------------------------------\n");
    printf("CPU Time:             %.2f ms\n", cpu_time);
    printf("GPU Time:             %.2f ms\n", metrics.total_time);
    printf("Speedup:              %.2fx\n", cpu_time / metrics.total_time);
    printf("\nCompute Performance:\n");
    printf("CPU GFLOP/s:          %.2f\n", cpu_gflops);
    printf("GPU GFLOP/s:          %.2f\n", metrics.gflops);
    printf("Performance Ratio:     %.2fx\n", metrics.gflops / cpu_gflops);
    printf("============================================\n");
    
    return 0;
}
