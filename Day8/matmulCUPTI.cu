#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <nvperf_host.h>
#include <nvperf_cuda_host.h>
#include <nvperf_target.h>
#include <string.h>

// Matrix dimensions 
const int M = 4092;
const int K = 4092;
const int N = 4092;

// Add error checking macro at top
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// Add error checking macro for CUPTI
#define CHECK_CUPTI(err) \
    do { \
        CUptiResult _err = (err); \
        if (_err != CUPTI_SUCCESS) { \
            const char *errstr; \
            cuptiGetResultString(_err, &errstr); \
            fprintf(stderr, "CUPTI error at %s:%d: %s\n", __FILE__, __LINE__, errstr); \
            exit(1); \
        } \
    } while(0)

// Update to use performance API metrics
const char* METRIC_NAMES[] = {
    "sm__cycles_active.avg.pct_of_peak_sustained_elapsed",     // SM utilization
    "dram__bytes.sum.per_second",                             // Memory bandwidth
    "sm__sass_thread_inst_executed_op_dfma_pred_on.sum"       // FMA instructions
};
const int NUM_METRICS = sizeof(METRIC_NAMES)/sizeof(METRIC_NAMES[0]);

dim3 grid((N + 32 - 1) / 32, (M + 32 - 1) / 32, 1);
dim3 block(32, 32, 1);

__global__ void matmul_naive(int M, int N, int K, const float *A,
                            const float *B, float *C) {
    int col_c = blockDim.x * blockIdx.x + threadIdx.x;
    int row_c = blockDim.y * blockIdx.y + threadIdx.y;

    if (col_c < N && row_c < M){
        float accu = 0.0f;
        for (int sum_index = 0; sum_index < K; sum_index+=1){
            accu += A[row_c * K + sum_index] * B[sum_index * N + col_c];
        }
        C[row_c * N + col_c] = accu;
    }
}

bool verifyResults(float* C_gpu, const float* C_cpu, int M, int N) {
    const float epsilon = 1e-2;
    for(int i = 0; i<M*N; i++) {
        if(abs(C_gpu[i] - C_cpu[i]) > epsilon) {
            printf("Verification failed at index %d: GPU=%f, CPU=%f\n", i, C_gpu[i], C_cpu[i]);
            return false;
        }
    }
    return true;
}

void initializeMatrices(float* A, float* B, int M, int K, int N) {
    // Corrected parameter order in initialization
    // initialize A to random 
    for (int i=0; i<M*K; i++){
        A[i] = rand() / (float)RAND_MAX;
    }

    // initialize B to identity
    for (int i=0; i<K*N; i++){
        if (i % (N+1) == 0){
            B[i] = 1;
        } else {
            B[i] = 0;
        }
    }
}

// Performance metrics structure
struct PerformanceMetrics {
    float kernel_time;
    float memory_time;
    float total_time;
    float gflops;
    float sm_efficiency;
    float dram_throughput;
    float tensor_core_util;
    bool correct;
};

// CUPTI initialization function
void initCupti() {
    CUpti_SubscriberHandle subscriber;
    CHECK_CUPTI(cuptiSubscribe(&subscriber, 
        [](void* userdata, CUpti_CallbackDomain domain, 
           CUpti_CallbackId cbid, const void* cbdata) {}, 
        nullptr));
}

typedef void (*KernelFunction)(int, int, int, const float*, const float*, float*);

PerformanceMetrics runKernel(KernelFunction kernel, const char* name, 
                            const float* h_A, const float* d_A, 
                            const float* d_B, float* d_C) 
{
    PerformanceMetrics metrics = {0};
    float* h_C = (float*)malloc(M * N * sizeof(float));

    // Initialize NVPW
    NVPW_InitializeHost_Params initializeHostParams = {NVPW_InitializeHost_Params_STRUCT_SIZE};
    NVPW_InitializeHost(&initializeHostParams);

    // Get current context and device
    hipCtx_t context;
    hipDevice_t device;
    hipCtxGetCurrent(&context);
    hipCtxGetDevice(&device);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);

    // Run kernel
    kernel<<<grid, block>>>(M, N, K, d_A, d_B, d_C);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&metrics.kernel_time, start, stop);

    // Calculate theoretical metrics
    float operations = 2.0f * M * N * K;  // multiply-add counts as 2 operations
    metrics.gflops = (operations / 1e9) / (metrics.kernel_time / 1000.0f);

    // Calculate memory throughput
    size_t bytes_read = M * K * sizeof(float) + K * N * sizeof(float);  // Reading A and B
    size_t bytes_written = M * N * sizeof(float);  // Writing to C
    float total_gb = (bytes_read + bytes_written) / (float)(1024*1024*1024);
    metrics.dram_throughput = total_gb / (metrics.kernel_time / 1000.0f);  // GB/s

    // Estimate SM efficiency based on occupancy
    int maxThreadsPerMultiProcessor;
    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerMultiProcessor, 
                          hipDeviceAttributeMaxThreadsPerMultiProcessor, 
                          0);
    hipDeviceGetAttribute(&maxThreadsPerBlock,
                          hipDeviceAttributeMaxThreadsPerBlock,
                          0);
    
    int numBlocks = grid.x * grid.y;
    int threadsPerBlock = block.x * block.y;
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
    
    // Calculate theoretical maximum blocks per SM
    int maxBlocksPerSM = maxThreadsPerMultiProcessor / threadsPerBlock;
    
    // Calculate blocks per SM (limited by both hardware and grid size)
    int blocksPerSM = min(numBlocks / numSMs, maxBlocksPerSM);
    
    // Calculate active threads per SM
    int activeThreadsPerSM = blocksPerSM * threadsPerBlock;
    
    // Calculate occupancy as a percentage
    metrics.sm_efficiency = (float)activeThreadsPerSM / maxThreadsPerMultiProcessor;

    // Clamp efficiency to 100%
    metrics.sm_efficiency = min(metrics.sm_efficiency, 1.0f);

    // Copy results back and verify
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    metrics.correct = verifyResults(h_C, h_A, M, N);

    // Print metrics
    printf("\n=== Performance Metrics (%s) ===\n", name);
    printf("Kernel Execution Time: %.3f ms\n", metrics.kernel_time);
    printf("GFLOPS: %.2f\n", metrics.gflops);
    printf("SM Efficiency (Occupancy): %.2f%%\n", metrics.sm_efficiency * 100.0f);
    printf("Memory Throughput: %.2f GB/s\n", metrics.dram_throughput);
    printf("Correctness: %s\n", metrics.correct ? "PASS" : "FAIL");

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_C);

    return metrics;
}

int main() {
    // Initialize CUDA driver API
    hipInit(0);
    
    // define host pointers 
    float *h_A, *h_B, *h_C;
    // define device pointers 
    float *d_A, *d_B, *d_C;

    // initialise matrices sizes 
    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    // allocate memory on the CPU
    h_A = (float*)malloc(size_a);
    h_B = (float*)malloc(size_b);
    h_C = (float*)malloc(size_c);

    // allocate memory on GPU
    CHECK_CUDA(hipMalloc((void**)&d_A, size_a));
    CHECK_CUDA(hipMalloc((void**)&d_B, size_b));
    CHECK_CUDA(hipMalloc((void**)&d_C, size_c));

    initializeMatrices(h_A, h_B, M, K, N);
    printf("init matrices");
    // send data to GPU 
    CHECK_CUDA(hipMemcpy(d_A, h_A, size_a, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size_b, hipMemcpyHostToDevice));

    printf("Going to run kernel\n");  // Add newline for proper output
    PerformanceMetrics metrics = runKernel(matmul_naive, "matmul_naive", h_A, d_A, d_B, d_C);
    printf("Done running kernel\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
}
